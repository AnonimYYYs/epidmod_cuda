#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <ctime>
#include <vector>
#include <algorithm>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/random/normal_distribution.h>

__global__ void init(
	int *state,
	bool *isUpdate,
	float *chance,
	thrust::minstd_rand *gen,
	thrust::uniform_real_distribution<float> *randFloat,
	thrust::uniform_int_distribution<int> *randPeople,
	thrust::normal_distribution<float> *randRemoved,
	thrust::normal_distribution<float> *randInfected,
	unsigned int *connAddr,
	unsigned int *connSize,
	unsigned int *conns,
	int peopleAmount,
	int totalThreads,
	unsigned int connOneSide,
	float infPart,
	float expPart,
	time_t seed
){
	int startIndex = threadIdx.x + blockIdx.x * blockDim.x;
	for(int i = startIndex; i < peopleAmount; i += totalThreads)
	{
		gen[i] = thrust::minstd_rand(seed + i * 10000);
		randFloat[i] = thrust::uniform_real_distribution<float>(0, 1);
		randPeople[i] = thrust::uniform_int_distribution<int>(0, peopleAmount - 1);
		randRemoved[i] = thrust::normal_distribution<float>(13.5f, 6.0f);
		randInfected[i] = thrust::normal_distribution<float>(5.5f, 3.0f);
		isUpdate[i] = false;

		// init states
		state[i] = 0;
		chance[i] = (randFloat[i])(gen[i]);
		if (i < (int)((expPart + infPart) * peopleAmount)) 
		{    // exposed
			(state[i])++;
			chance[i] = (randInfected[i])(gen[i]);
		}
		if (i < (int)(infPart * peopleAmount)) 
		{                                // infectious
			(state[i])++;
			chance[i] = (randRemoved[i])(gen[i]);
		}

		// init graph
		connAddr[i] = i * connOneSide * 2;
		connSize[i] = connOneSide * 2;
		for(int j = 0; j < connOneSide; j++)
		{
			conns[connAddr[i] + j] = (i + j + 1) % peopleAmount;
			conns[connAddr[i] + connOneSide + j] = (i - j - 1 + peopleAmount) % peopleAmount;
		}
	}
}

__global__ void calcUpdStates(
	int *state,
	bool *isUpdate,
	float *chance,
	thrust::minstd_rand *gen,
	thrust::uniform_int_distribution<int> *randPeople,
	unsigned int *connAddr,
	unsigned int *connSize,
	unsigned int *conns,
	unsigned int *randConns,
	int randConnsAmount,
	int peopleAmount,
	int totalThreads,
	float infChance
){
	int startIndex = threadIdx.x + blockIdx.x * blockDim.x;
	for(int i = startIndex; i < peopleAmount; i += totalThreads)
	{// calc chances
		if(state[i] == 0)
		{
			for(unsigned int rndInd = i * randConnsAmount; rndInd < (i + 1) * randConnsAmount; rndInd++)
			{// add random edges
				randConns[rndInd] = (randPeople[i])(gen[i]);
			}

			//calc chance
			float ch = 1.0f;
			for(unsigned int ind = connAddr[i]; ind < connAddr[i] + connSize[i]; ind++)
			{
				if(state[conns[ind]] == 2) { ch *= 1 - infChance; }
			}
			for(unsigned int rndInd = i * randConnsAmount; rndInd < (i + 1) * randConnsAmount; rndInd++)
			{
				bool isAdd = true;
				for(unsigned int ind = connAddr[i]; ind < connAddr[i] + connSize[i]; ind++)
				{
					if(conns[ind] == randConns[rndInd]) { isAdd = false; break; }
				}
				if(isAdd)
				{
					for(unsigned int j = i * randConnsAmount; j < rndInd; j++)
					{
						if(randConns[j] == randConns[rndInd]) { isAdd = false; break; }
					}

					if(isAdd)
					{
						if(state[randConns[rndInd]] == 1) { ch *= 1 - expChance; }
						if(state[randConns[rndInd]] == 2) { ch *= 1 - infChance; }
					}
				}
			}
			//check chance
			if(ch < chance[i]) { isUpdate[i] = true; }

		}

		if(state[i] == 1 || state[i] == 2)
		{
			chance[i]--;
			if(chance[i] <= 0) { isUpdate[i] = true; }
		}

	}
}


__global__ void updateStates(
	int *state,
	bool *isUpdate,
	float *chance,
	thrust::minstd_rand *gen,
	thrust::uniform_real_distribution<float> *randFloat,
	thrust::normal_distribution<float> *randRemoved,
	thrust::normal_distribution<float> *randInfected,
	int peopleAmount,
	int totalThreads
){
	int startIndex = threadIdx.x + blockIdx.x * blockDim.x;
	for(int i = startIndex; i < peopleAmount; i += totalThreads)
	{
		if(isUpdate[i])
		{
			if(state[i] == 0) { chance[i] = (randInfected[i])(gen[i]); }
			if(state[i] == 1) { chance[i] = (randRemoved[i])(gen[i]); }
			state[i]++;
			isUpdate[i] = false;
		}

		if(state[i] == 0) { chance[i] = (randFloat[i])(gen[i]); }
	}
}



int main()
{

	printf("watts-strogaz reroll with vectors, cuda\n");
	// declaring variables , do changes here vvv
	int threads = 64;
	int blocks = 64;

	unsigned int peopleAmount = 300000;

	unsigned int connAmount = 6;
	unsigned int randConnsAmount = 6;

	float infectedPart = 0.05f;
	float exposedPart = 0.1f;

	float wsgk = 0.2f;

	float infCatch = 0.0558f;

	unsigned int repeatTime = 50; // days

	unsigned int repeats = 500; // loops for time counting
	// no more changes from here ^^^^

	int totalThreads = threads * blocks;

	unsigned int connOneSide = connAmount / 2;

	thrust::minstd_rand *devGen;
	thrust::uniform_real_distribution<float> *devRandFloat;
	thrust::uniform_int_distribution<int> *devRandPeople;
	thrust::normal_distribution<float> *devRandRemoved;
	thrust::normal_distribution<float> *devRandInfected;

	int *state, *devState;
	bool *devIsUpdate;
	float *devChance;
	unsigned int *connAddr,*devConnAddr, *connSize, *devConnSize, *conns, *devConns;
	unsigned int *devRandConns;

	// memory allocating
	connAddr = (unsigned int*)malloc(sizeof(unsigned int) * peopleAmount);
	connSize = (unsigned int*)malloc(sizeof(unsigned int) * peopleAmount);
	conns = (unsigned int*)malloc(sizeof(unsigned int) * peopleAmount * connOneSide * 2);
	hipMalloc((void**)&devConnAddr, sizeof(unsigned int) * peopleAmount);
	hipMalloc((void**)&devConnSize, sizeof(unsigned int) * peopleAmount);
	hipMalloc((void**)&devConns, sizeof(unsigned int) * peopleAmount * connOneSide * 2);

	hipMalloc((void**)&devRandConns, sizeof(unsigned int) * peopleAmount * randConnsAmount);

	state = (int*)malloc(sizeof(int) * peopleAmount);
	hipMalloc((void**)&devState, sizeof(int) * peopleAmount);
	hipMalloc((void**)&devIsUpdate, sizeof(bool) * peopleAmount);
	hipMalloc((void**)&devChance, sizeof(float) * peopleAmount);
	hipMalloc((void**)&devGen, sizeof(thrust::minstd_rand) * peopleAmount);
	hipMalloc((void**)&devRandFloat, sizeof(thrust::uniform_real_distribution<float>) * peopleAmount);
	hipMalloc((void**)&devRandPeople, sizeof(thrust::uniform_int_distribution<int>) * peopleAmount);
	hipMalloc((void**)&devRandRemoved, sizeof(thrust::normal_distribution<float>) * peopleAmount);
	hipMalloc((void**)&devRandInfected, sizeof(thrust::normal_distribution<float>) * peopleAmount);

	// do things
	time_t begin = time(0);
	printf("people: %u\n", peopleAmount);
	for(unsigned int n = 0; n < repeats; n++)
	{

		time_t timeSeed = time(0);
		// initialize variables
		init<<<blocks,threads>>>(
			devState,
			devIsUpdate,
			devChance,
			devGen,
			devRandFloat,
			devRandPeople,
			devRandRemoved,
			devRandInfected,
			devConnAddr,
			devConnSize,
			devConns,
			peopleAmount,
			totalThreads,
			connOneSide,
			infectedPart,
			exposedPart,
			timeSeed
		);

		// watts-strogaz reroll
		hipMemcpy(connAddr, devConnAddr, sizeof(unsigned int) * peopleAmount, hipMemcpyDeviceToHost);
		hipMemcpy(connSize, devConnSize, sizeof(unsigned int) * peopleAmount, hipMemcpyDeviceToHost);
		hipMemcpy(conns, devConns, sizeof(unsigned int) * peopleAmount * connOneSide * 2, hipMemcpyDeviceToHost);

		thrust::minstd_rand gen(timeSeed - 10000);
		thrust::uniform_real_distribution<float> randFloat(0, 1);
		thrust::uniform_int_distribution<int> randPeople(0, peopleAmount - 1);

		// vector realisation of watts-strogatz
		std::vector<unsigned int>  vecConns[peopleAmount];
		for(unsigned int i = 0; i < peopleAmount; i++)
		{
			std::vector<unsigned int> toAdd = std::vector<unsigned int>();
			for(unsigned int j = connAddr[i]; j < connAddr[i] + connSize[i]; j++) { toAdd.push_back(conns[j]); }
			vecConns[i] = toAdd;
		}

		for(unsigned int i = 0; i < peopleAmount; i++)
		{
			for (unsigned int ind = 0; ind < vecConns[i].size(); ind++)
			{
				if (randFloat(gen) < wsgk)
				{       // if edge need to be replaced
					unsigned int new_adr;
					do { new_adr = randPeople(gen); } while (new_adr == i);         // new edge without connecting vertex to itself
					if (std::find(vecConns[new_adr].begin(), vecConns[new_adr].end(), i) == vecConns[new_adr].end())
					{               // if edge not exist
						vecConns[new_adr].push_back(i);         // adding edge to 'new' vertex
						vecConns[vecConns[i][ind]].erase(std::find(vecConns[vecConns[i][ind]].begin(), vecConns[vecConns[i][ind]].end(), i));           // remove edge from 'old' vertex
						vecConns[i][ind] = new_adr;     // change edge to initial vertex
					}
				}
			}
		}
		unsigned int ind = 0;
		for(unsigned int i = 0; i < peopleAmount; i++)
		{
			connAddr[i] = ind;
			connSize[i] = vecConns[i].size();
			for(unsigned int j = 0; j < vecConns[i].size(); j++) { conns[ind] = vecConns[i][j]; }
		}
		hipMemcpy(devConnAddr, connAddr, sizeof(unsigned int) * peopleAmount, hipMemcpyHostToDevice);
		hipMemcpy(devConnSize, connSize, sizeof(unsigned int) * peopleAmount, hipMemcpyHostToDevice);
		hipMemcpy(devConns, conns, sizeof(unsigned int) * peopleAmount * connOneSide * 2, hipMemcpyHostToDevice);

		// end of watts-strogaz

		for(unsigned int t = 0; t < repeatTime; t++)
		{// updating states
			calcUpdStates<<<blocks,threads>>>(
				devState,
				devIsUpdate,
				devChance,
				devGen,
				devRandPeople,
				devConnAddr,
				devConnSize,
				devConns,
				devRandConns,
				randConnsAmount,
				peopleAmount,
				totalThreads,
				infCatch
			);

			updateStates<<<blocks,threads>>>(
				devState,
				devIsUpdate,
				devChance,
				devGen,
				devRandFloat,
				devRandRemoved,
				devRandInfected,
				peopleAmount,
				totalThreads
			);
		}

		hipMemcpy(state, devState, sizeof(int) * peopleAmount, hipMemcpyDeviceToHost);

		int s0 = 0, s1 = 0, s2 = 0, s3 = 0;
		for(int i = 0; i < peopleAmount; i++) {
		if(state[i] == 0) { s0++; }
		if(state[i] == 1) { s1++; }
		if(state[i] == 2) { s2++; }
		if(state[i] == 3) { s3++; }
		}

		printf("%u:\t%i\t%i\t%i\t%i\n", n, s0, s1, s2, s3);
	}
	time_t end = time(0);
	printf("%llu\n%f\n", end - begin, static_cast<float>(end - begin) / static_cast<float>(repeats));

	// memory free
	hipFree(devState);
	hipFree(devIsUpdate);
	hipFree(devChance);
	hipFree(devGen);
	hipFree(devRandFloat);
	hipFree(devRandPeople);
	hipFree(devRandRemoved);
	hipFree(devRandInfected);
	hipFree(devConnAddr);
	hipFree(devConnSize);
	hipFree(devConns);
	free(connAddr);
	free(connSize);
	free(conns);
	free(state);

	return(0);
}
